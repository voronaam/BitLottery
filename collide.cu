#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "helper_timer.h"
#include "secp256k1.h"
#include "secp256k1_preallocated.h"
#include "group.h"
#include "ripemd160.h"
#include "dev_sha256.h"
#include "dev_key.h"
#include "keccak256.cu"
#include "targets2.h"

#define THREADS 32*6
#define KERNEL_ITERATIONS 1000000
#define HOST_ITERATIONS 1*1000 // 1 is about 6 min on a consumer grade laptop (Dell XPS)

#define TICKETS THREADS*THREADS // Should be THREADS**2

__device__ static int inline compare_dir(const unsigned char *a, const unsigned char *b) {
    #pragma unroll 20
    for (int i = 0; i < 20; i++) {
        if (a[i] > b[i]) {
            return 1;
        } else if (a[i] < b[i]) {
            return -1;
        }
    }
    return 0;
}

// Just a regular binary search. From glibc.
__device__ static unsigned char *bsearch_dev(const unsigned char *__key,
                                      const unsigned char *__base,
                                      size_t __nmemb, size_t __size) {
  size_t __l, __u, __idx;
  const unsigned char *__p;
  int __comparison;

  __l = 0;
  __u = __nmemb;
  while (__l < __u) {
    __idx = (__l + __u) / 2;
    __p = (const unsigned char *)(((const unsigned char *)__base) +
                                  (__idx * __size));
    __comparison = compare_dir(__key, __p);
    if (__comparison < 0)
      __u = __idx;
    else if (__comparison > 0)
      __l = __idx + 1;
    else {
      return (unsigned char *)__p;
    }
  }

  return NULL;
}

/// 128 bytes for perfect alignment
typedef struct {
    unsigned char seckey[32];
    rustsecp256k1_v0_10_0_ge p;
    uint64_t offset;
} lottery_ticket;

__global__ void incKernel(lottery_ticket *g_idata, lottery_ticket *g_odata) {
    int i;
    unsigned char serialized_pubkey[65], hash1[32], hash2[20], hash3[32];
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    #pragma unroll 4
    for (i = 0; i < KERNEL_ITERATIONS; i++, dev_inc(&g_idata[index].p)) {
        // Make first byte of serialized_pubkey to be the oddity tag and bytes 1-33 to be the x coordinate
        dev_eckey_pubkey_serialize(&g_idata[index].p, serialized_pubkey);
        // Make bytes 34-65 to be the y coordinate
        dev_fe_impl_get_b32(serialized_pubkey + 33, &g_idata[index].p.y);

        sha256(hash1, serialized_pubkey); // Hash first 33 bytes
        ripemd160(hash1, 32, hash2); // hash2 is the BTC target now

        Keccak256_getHash(serialized_pubkey + 1, 64, hash3); // last 20 bytes of hash3 is the ETH target now

        unsigned char* btc = (unsigned char*) bsearch_dev (hash2, (unsigned char*)targets2btc, MAX_BTC_TARGETS2, 20);
        unsigned char* eth = (unsigned char*) bsearch_dev (hash3 + 12, (unsigned char*)targets2eth, MAX_ETH_TARGETS2, 20);
        if (btc != NULL || eth != NULL) {
            printf("Found a match! Offset from PK: %lx\n", g_idata[index].offset + i);
            for (int j = 0; j < 32; j++) {
                printf("%02x", g_idata[index].seckey[j]);
            }
            printf("\nPrinting the combined pubkey\n ±<...........................x..................................><............................y.................................>\n");
            for (int j = 0; j < 65; j++) {
                printf("%02x", serialized_pubkey[j]);
            }
            printf("\n");
            if (btc != NULL) {
                printf("BTC target: ");
                for (int j = 0; j < 20; j++) {
                    printf("%02x", btc[j]);
                }
                printf("\n");
            }
            if (eth != NULL) {
                printf("ETH target: ");
                for (int j = 0; j < 20; j++) {
                    printf("%02x", eth[j]);
                }
                printf("\n");
            }
        }
    }

    memcpy(g_odata[index].seckey, g_idata[index].seckey, 32);
    g_odata[index].p = g_idata[index].p;
    g_odata[index].offset = g_idata[index].offset + i;
}

// Use the full secp256k1 library to create the initial keypairs
static void create_tickets(lottery_ticket *tickets) {
    size_t ctx_size = rustsecp256k1_v0_10_0_context_preallocated_size(SECP256K1_CONTEXT_NONE); // 208
    void *ctx_mem = malloc(ctx_size);
    rustsecp256k1_v0_10_0_context *ctx = rustsecp256k1_v0_10_0_context_preallocated_create(ctx_mem, SECP256K1_CONTEXT_NONE);

    // Fill the tickets with keypairs
    for (int i = 0; i < TICKETS; i++) {
        // random 32 bytes of secret key
        for (int j = 0; j < 32; j++) {
            tickets[i].seckey[j] = rand() % 256;
        }
        expose_pubkey_create(ctx, &tickets[i].p, tickets[i].seckey);
    }

    free(ctx_mem);
}

int main(int argc, char **argv) {
    lottery_ticket tickets[TICKETS] = { 0 };
    create_tickets(tickets);

    int devID = findCudaDevice(argc, (const char **)argv); // use command-line specified CUDA device, otherwise use device with highest Gflops/s

    unsigned int num_threads = TICKETS;
    unsigned int mem_size = sizeof(lottery_ticket) * TICKETS;
    // allocate device memory
    lottery_ticket *d_idata;
    checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
    lottery_ticket *d_odata;
    checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));
    // setup execution parameters
    dim3 grid(THREADS);
    dim3 threads(THREADS);

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    for (int k = 0; k < HOST_ITERATIONS; k++) {
        checkCudaErrors(hipMemcpy(d_idata, &tickets, mem_size, hipMemcpyHostToDevice));
        incKernel<<<grid, threads, 0>>>(d_idata, d_odata);
        getLastCudaError("Kernel execution failed");
        checkCudaErrors(hipMemcpy(&tickets, d_odata, mem_size, hipMemcpyDeviceToHost));
    }

    sdkStopTimer(&timer);

    unsigned __int128 total = 0;
    for (int i = 0; i < TICKETS; i++) {
        total += tickets[i].offset;
    }
    printf("Total: %llu %llu in %fms\n", (unsigned long long)(total >> 64), (unsigned long long)total, sdkGetTimerValue(&timer));

    // cleanup memory
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    sdkDeleteTimer(&timer);

    return 0;
}
