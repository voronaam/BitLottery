#include <cstdint>
#include <cassert>
#include <hip/hip_runtime.h>

static constexpr int Keccak256_HASH_LEN = 32;

__device__ static void Keccak256_getHash(const uint8_t msg[], size_t len, uint8_t hashResult[Keccak256_HASH_LEN]);

__device__ static void Keccak256_absorb(uint64_t state[5][5]);
__device__ static uint64_t Keccak256_rotl64(uint64_t x, int i);

// #define UINT64_C(c) (c ## ULL)

__device__ constexpr int Keccak256_NUM_ROUNDS = 24;

__device__ constexpr unsigned char Keccak256_ROTATION[5][5] = {
    { 0, 36,  3, 41, 18},
    { 1, 44, 10, 45,  2},
    {62,  6, 43, 15, 61},
    {28, 55, 25, 21, 56},
    {27, 20, 39,  8, 14}
};

__device__ __forceinline__ uint64_t Keccak256_rotl64(uint64_t x, int i) {
    return ((0U + x) << i) | (x >> ((64 - i) & 63));
}

__device__ __forceinline__ void Keccak256_absorb(uint64_t state[5][5]) {
    uint64_t (*a)[5] = state;
    uint8_t r = 1;  // LFSR
    for (int i = 0; i < Keccak256_NUM_ROUNDS; i++) {
        // Theta step
        uint64_t c[5] = {};
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                c[x] ^= a[x][y];
        }
        for (int x = 0; x < 5; x++) {
            uint64_t d = c[(x + 4) % 5] ^ Keccak256_rotl64(c[(x + 1) % 5], 1);
            for (int y = 0; y < 5; y++)
                a[x][y] ^= d;
        }

        // Rho and pi steps
        uint64_t b[5][5];
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                b[y][(x * 2 + y * 3) % 5] = Keccak256_rotl64(a[x][y], Keccak256_ROTATION[x][y]);
        }

        // Chi step
        for (int x = 0; x < 5; x++) {
            for (int y = 0; y < 5; y++)
                a[x][y] = b[x][y] ^ (~b[(x + 1) % 5][y] & b[(x + 2) % 5][y]);
        }

        // Iota step
        for (int j = 0; j < 7; j++) {
            a[0][0] ^= static_cast<uint64_t>(r & 1) << ((1 << j) - 1);
            r = static_cast<uint8_t>((r << 1) ^ ((r >> 7) * 0x171));
        }
    }
}

__device__ __forceinline__ void Keccak256_getHash(const uint8_t msg[], size_t len, uint8_t hashResult[Keccak256_HASH_LEN]) {
    uint64_t state[5][5] = {};

    // XOR each message byte into the state, and absorb full blocks
    int blockOff = 0;
    const int BLOCK_SIZE = 200 - Keccak256_HASH_LEN * 2; // Define BLOCK_SIZE here
    for (size_t i = 0; i < len; i++) {
        int j = blockOff >> 3;
        state[j % 5][j / 5] ^= static_cast<uint64_t>(msg[i]) << ((blockOff & 7) << 3);
        blockOff++;
        if (blockOff == BLOCK_SIZE) {
            Keccak256_absorb(state);
            blockOff = 0;
        }
    }

    // Final block and padding
    {
        int i = blockOff >> 3;
        state[i % 5][i / 5] ^= UINT64_C(0x01) << ((blockOff & 7) << 3);
        blockOff = BLOCK_SIZE - 1;
        int j = blockOff >> 3;
        state[j % 5][j / 5] ^= UINT64_C(0x80) << ((blockOff & 7) << 3);
        Keccak256_absorb(state);
    }

    // Uint64 array to bytes in little endian
    for (int i = 0; i < Keccak256_HASH_LEN; i++) {
        int j = i >> 3;
        hashResult[i] = static_cast<uint8_t>(state[j % 5][j / 5] >> ((i & 7) << 3));
    }
}
